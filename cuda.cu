#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>

#define N 1024 * 220


__device__ float f(float x) 
{
    return sinf(x)*x+cosf(x*x)*x*x;
}

__global__ void kernel(float* data)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    float stepbythread = 100.f / (1024.f * 220.f); 
    float begin = idx * stepbythread;
    float end = begin + stepbythread;

    float x = begin;
    float cur_min = f(x);

    x += 0.0001f;
    while (x <= end)
    {
        float cur_f = f(x);

        if (cur_f < cur_min)
            cur_min = cur_f;

        x += 0.0001f;
    }

    data[idx] = cur_min;

    if (idx % 512 == 0)
    {
        float this_min = data[idx];
        for (int i = idx; i < idx + 512; i++)
        {
            if (data[idx] > data[i])
                data[idx] = data[i];
        }
    }
}

int main()
{
    int deviceCount;
    hipDeviceProp_t devProp;
    hipGetDeviceCount(&deviceCount);
    printf("------------------------------------------\n");
    printf("Found %d devices\n", deviceCount);
    for (int device = 0; device < deviceCount; device++)
    {
        hipGetDeviceProperties(&devProp, device);
        printf("Device                  : %d\n", device);
        printf("Compute copability      : %d.%d\n", devProp.major, devProp.minor);
        printf("Name                    : %s\n", devProp.name);
        printf("Total Global Memory     : %d\n", devProp.totalGlobalMem);
        printf("Shared memory per block : %d\n", devProp.sharedMemPerBlock);
        printf("Register per block      : %d\n", devProp.regsPerBlock);
        printf("Warp size               : %d\n", devProp.warpSize);
        printf("Shared memory per block : %d\n", devProp.sharedMemPerBlock);
        printf("Max pthreads per block  : %d\n", devProp.maxThreadsPerBlock);
        printf("Total constant memory   : %d\n", devProp.totalConstMem);
    }
    printf("------------------------------------------\n\n");

    float z[N];
    float* dev = NULL;

    if (hipMalloc((void**)&dev, (N * sizeof(float))))
        printf("hipMalloc error\n");

    kernel<<<dim3((N / 512), 1), dim3(512, 1)>>>(dev);

    if (hipMemcpy(z, dev, (N * sizeof(float)), hipMemcpyDeviceToHost))
        printf("hipMemcpy error\n");

    if (hipFree(dev))
        printf("hipFree error");

    int count = 0;
    float min = z[0];
    for (int idx = 0; idx < N; idx += 512) 
    {
        //printf("a[%d] = %.5f\n", idx, z[idx]);

        if (z[idx] < z[0])
            min = z[idx];
        count++;
    }

    printf("count of blocks             : %d\n", N/512);
    printf("count of threads per block  : %d\n", 512);
    printf("%d out of %d comparisons were made on the processor\n%d out of %d comparisons were made on the graphics card\n", count, N, N-count, N);
    printf("f = sin(x) * x + cos(x * x) * x * x\n");
    printf("min(f) = %.5f\n", min);

    return 0;
}
